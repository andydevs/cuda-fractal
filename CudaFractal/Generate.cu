// Includes
#include "Generate.cuh"
#include "FractalKernels.cuh"
#include "lodepng.h"

// CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

// Libraries
#include <iostream>
#include <string>
#include <ctime>

/**
 * Generate fractal image
 *
 * @param mbrot    true if generating mandelbrot set
 * @param cons     complex constant
 * @param scale    scale transformation complex
 * @param trans    translate transformation complex
 * @param cmap     colormap to generate with
 * @param width    width of the image
 * @param height   height of the image
 * @param filename name of file to save to
 * @param mnemonic used to identify generator job
 */
void generate(bool mbrot, hipFloatComplex cons, hipFloatComplex scale, hipFloatComplex trans, colormap cmap, unsigned width, unsigned height, std::string filename, std::string mnemonic) {
	DEFINE_TIMES

	// NOTE: 
	//	Investigate why grid spaces or block spaces 
	//	do not work in this case when made rectangular...

	// Create a cuda-managed image buffer and save location at image
	unsigned char* image;
	unsigned length = width*height*IMAGE_NUM_CHANNELS;
	hipMallocManaged(&image, sizeof(unsigned char)*length);

	// Where the magic happens...
	// Call CUDA kernel on the given grid space of blocks
	// Each block being a block space of threads.
	// Each thread computes a separate pixel in the Julia/mandelbrot set
	DOING("Running kernel for " + mnemonic);
	if (mbrot) { mandelbrotset_launcher(scale, trans, cmap, width, height, image); }
	else { juliaset_launcher(cons, scale, trans, cmap, width, height, image); }
	hipDeviceSynchronize(); // Wait for kernel to finish
	DONE();

	// Save img buffer to png file
	DOING("Saving png");
	lodepng_encode32_file(filename.c_str(), image, width, height);
	DONE();

	// Free image buffer and exit
	hipFree(image);
};