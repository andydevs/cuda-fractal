#include "hip/hip_runtime.h"
#ifndef __FRACTAL_HOST_CODE__
#define __FRACTAL_HOST_CODE__

// Includes
#include "DeviceCode.cuh"
#include "lodepng.h"
#include <hip/hip_runtime.h>
#include <boost\program_options.hpp>

// Libraries
#include <exception>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <cstdio>
#include <cmath>
#include <ctime>
#include <map>

// Start time
clock_t start;

// Macros
#define DOING(task) \
	std::cout << task << "..."; \
	start = clock();
#define DONE() \
	std::cout << "Done! " \
		<< (float)(clock() - start) / CLOCKS_PER_SEC << "s" \
		<< std::endl;

// Boost namespaces
namespace po = boost::program_options;

/**
 * Returns the preset colormap of the given name
 *
 * @param name the name of the colormap
 * 
 * @return the preset colormap
 */
colormap fromPreset(std::string name) {
	// Presets map
	std::map<std::string, colormap> presets;

	// Populate presets map
	presets["blackwhite"] = colormap::gradient(
		color::hex(0x000000), 
		color::hex(0xffffff));
	presets["nvidia"] = colormap::gradient(
		color::hex(0x000000),
		color::hex(0xa3ff00));
	presets["saffron"] = colormap::sinusoid(
		fColor(1.4, 1.4, 1.4),
		fColor(-2.0, -3.0, -4.0),
		0xff);

	// Return appropriate preset
	return presets[name];
};


/**
 * The main procedure
 *
 * @param argc the number of command line args
 * @param argv the command line args
 *
 * @return status code
 */
int main(int argc, const char* argv[]) {
	// Soon-to-be user-inputted data
	bool help, mbrot;
	float consr, consi;
	unsigned width, height;
	std::string cname, fname;

	// Get user input
	po::options_description options("> CUDAFractal [options]");
	options.add_options()
		("help", po::bool_switch(&help), "print help message")
		("mbrot", po::bool_switch(&mbrot), "compute the mandelbrot fractal algorithm")
		("cr", po::value<float>(&consr)->default_value(-0.4), "real value of c")
		("ci", po::value<float>(&consi)->default_value(0.6), "imaginary value of c")
		("width", po::value<unsigned>(&width)->default_value(1920), "image width")
		("height", po::value<unsigned>(&height)->default_value(1080), "image height")
		("cmap", po::value<std::string>(&cname)->default_value("nvidia"), "colormap preset")
		("file", po::value<std::string>(&fname), "output file name");
	po::variables_map vars;
	po::store(po::parse_command_line(argc, argv, options), vars);
	po::notify(vars);

	// Exit if no filename specified!
	if (fname.empty()) {
		std::cout << "ERROR: No filename specified!" << std::endl;
		return 1;
	}

	// Get colormap and constant
	colormap cmap = fromPreset(cname);
	hipFloatComplex cons = make_hipFloatComplex(consr, consi);

	// Block space
	// Using 8x8 thread block space because that 
	// divides evenly into most standard resolutions
	int blockSize = 8;
	dim3 blockSpace(blockSize, blockSize);
		
	// Grid space
	// Find the largest side of the image rectangle 
	// and make a square out of that side. Divide 
	// number oftotal "threads" by the block size. 
	// This is the number of the blocks in the grid
	int gridSize = (width >= height ? width : height) / blockSize;
	dim3 gridSpace(gridSize, gridSize);

	// NOTE: 
	//	Investigate why grid spaces or block spaces 
	//	do not work in this case when made rectangular...

	// Create a cuda-managed image buffer and save location at image
	unsigned char* image;
	unsigned length = width*height*IMAGE_NUM_CHANNELS;
	hipMallocManaged(&image, sizeof(unsigned char)*length);

	// Where the magic happens...
	// Call CUDA kernel on the given grid space of blocks
	// Each block being a block space of threads.
	// Each thread computes a separate pixel in the Julia/mandelbrot set
	if (mbrot) {
		DOING("Running Mandelbrot set kernel");
		mandelbrotset<<<gridSpace, blockSpace>>>(cmap, width, height, image);
		hipDeviceSynchronize(); // Wait for kernel to finish
		DONE();
	}
	else {
		DOING("Running Julia set kernel");
		juliaset<<<gridSpace, blockSpace>>>(cons, cmap, width, height, image);
		hipDeviceSynchronize(); // Wait for kernel to finish
		DONE();
	}

	// Save img buffer to png file
	DOING("Saving png");
	lodepng_encode32_file(fname.c_str(), image, width, height);
	DONE();
	
	// Free image buffer and exit
	hipFree(image);
	return 0;
}

#endif // !__FRACTAL_HOST_CODE__