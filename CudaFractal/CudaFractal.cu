#include "hip/hip_runtime.h"
#ifndef __FRACTAL_HOST_CODE__
#define __FRACTAL_HOST_CODE__

// Includes
#include "DeviceCode.cuh"
#include "lodepng.h"
#include <hip/hip_runtime.h>
#include <boost\program_options.hpp>
#include <boost\property_tree\ptree.hpp>

// Libraries
#include <exception>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <cstdio>
#include <cmath>
#include <ctime>
#include <map>

// Start time
clock_t start;

// Macros
#define DOING(task) \
	std::cout << task << "..."; \
	start = clock();
#define DONE() \
	std::cout << "Done! " \
		<< (float)(clock() - start) / CLOCKS_PER_SEC << "s" \
		<< std::endl;

// Boost namespaces
namespace po = boost::program_options;
namespace pt = boost::property_tree;

// --------------------------------- PRESET PARSE ---------------------------------

// Presets map
bool uninitialized = true;
std::map<std::string, colormap> presets;

/**
 * Initializes the presets map
 */
void initPresets() {
	// Initialize if uninitialized
	if (uninitialized) {
		// Populate presets map
		presets["noir"] = colormap::gradient(
			color::hex(0x000000),
			color::hex(0xffffff));
		presets["ink"] = colormap::gradient(
			color::hex(0xffffff),
			color::hex(0x000000));
		presets["nvidia"] = colormap::gradient(
			color::hex(0x000000),
			color::hex(0xa3ff00));
		presets["orchid"] = colormap::gradient(
			color::hex(0xeeeeff),
			color::hex(0xff0000));
		presets["flower"] = colormap::sinusoid(
			fColor(0.7, 0.7, 0.7),
			fColor(-2.0, -2.0, -1.0));
		presets["psychedelic"] = colormap::sinusoid(
			fColor(5.0, 5.0, 5.0),
			fColor(4.1, 4.5, 5.0));
		presets["ice"] = colormap::sinusoid(
			fColor(2.0, 2.0, 0.1),
			fColor(0.0, 0.0, 2.0));
		presets["fruity"] = colormap::sinusoid(
			fColor(5.0, 5.0, 5.0),
			fColor(0.0, 4.5, 2.5));
		presets["sarree"] = colormap::sinusoid(
			fColor(1.4, 1.4, 1.4),
			fColor(2.0, 3.0, 4.0));
		presets["sarree2"] = colormap::sinusoid(
			fColor(1.00, 2.00, 2.00),
			fColor(F_P1, F_P1, F_P1));
		presets["lightgarden"] = colormap::sinusoid(
			fColor(1.00, 2.00, 9.00),
			fColor(F_N1, F_N1, F_N1));
		presets["acid"] = colormap::sinusoid(
			fColor(8.00, 9.00, 0.00),
			fColor(F_N1, F_N1, F_N1));

		// Toggle uninitialized
		uninitialized = false;
	}
};

/**
 * Returns the preset colormap of the given name
 *
 * @param name the name of the colormap
 * 
 * @return the preset colormap
 */
colormap fromPreset(std::string name) {
	initPresets();
	return presets[name];
};

/**
 * Lists all presets available
 */
void listPresets() {
	initPresets();
	std::cout << "Presets Available:" << std::endl;
	for each (std::pair<std::string, colormap> entry in presets) {
		std::cout << "    " << entry.first << std::endl;
	}
};

// -------------------------------- GENERATOR CODE --------------------------------

/**
 * Returns scale complex which incorporates rotation and zooming
 *
 * @param rotate the rotation value (in degrees)
 * @param zoom   the zoom value
 *
 * @return scale complex
 */
hipFloatComplex make_cuScaleComplex(float rotate, float zoom) {
	return make_hipFloatComplex(
		cos(rotate*F_PI / 180.0f) / zoom,
		sin(rotate*F_PI / 180.0f) / zoom);
};

/**
 * Generate fractal image
 *
 * @param mbrot    true if generating mandelbrot set
 * @param cons     complex constant
 * @param scale    scale transformation complex
 * @param trans    translate transformation complex
 * @param cmap     colormap to generate with
 * @param width    width of the image
 * @param height   height of the image
 * @param filename name of file to save to
 * @param mnemonic used to identify generator job
 */
void generate(bool mbrot, hipFloatComplex cons, hipFloatComplex scale, hipFloatComplex trans, colormap cmap, unsigned width, unsigned height, std::string filename, std::string mnemonic) {
	// Block space
	// Using 8x8 thread block space because that 
	// divides evenly into most standard resolutions
	int blockSize = 8;
	dim3 blockSpace(blockSize, blockSize);

	// Grid space
	// Find the largest side of the image rectangle 
	// and make a square out of that side. Divide 
	// number oftotal "threads" by the block size. 
	// This is the number of the blocks in the grid
	int gridSize = (width >= height ? width : height) / blockSize;
	dim3 gridSpace(gridSize, gridSize);

	// NOTE: 
	//	Investigate why grid spaces or block spaces 
	//	do not work in this case when made rectangular...

	// Create a cuda-managed image buffer and save location at image
	unsigned char* image;
	unsigned length = width*height*IMAGE_NUM_CHANNELS;
	hipMallocManaged(&image, sizeof(unsigned char)*length);

	// Where the magic happens...
	// Call CUDA kernel on the given grid space of blocks
	// Each block being a block space of threads.
	// Each thread computes a separate pixel in the Julia/mandelbrot set
	DOING("Running kernel for " + mnemonic);
	if (mbrot) { mandelbrotset << <gridSpace, blockSpace >> > (scale, trans, cmap, width, height, image); }
	else { juliaset << <gridSpace, blockSpace >> > (cons, scale, trans, cmap, width, height, image); }
	hipDeviceSynchronize(); // Wait for kernel to finish
	DONE();

	// Save img buffer to png file
	DOING("Saving png");
	lodepng_encode32_file(filename.c_str(), image, width, height);
	DONE();

	// Free image buffer and exit
	hipFree(image);
};

// ---------------------------------- XML PARSE -----------------------------------

/**
 * Executes job described in property tree
 *
 * @param job the job tree
 */
void doFractalJob(pt::ptree job) {
	// Get values from xml job tree
	std::string mnemonic = job.get("<xmlattr>.mnemonic", "xmlfractal");
	bool mbrot = job.get("<xmlattr>.mandelbrot", false);
	hipFloatComplex cons = make_hipFloatComplex(
		job.get("constant.<xmlattr>.real", -0.4f), 
		job.get("constant.<xmlattr>.imag", 0.6f));
	hipFloatComplex scale = make_cuScaleComplex(
		job.get("scale.<xmlattr>.rotate", 0.0f),
		job.get("scale.<xmlattr>.zoom", 1.0f));
	hipFloatComplex trans = make_hipFloatComplex(
		job.get("translate.<xmlattr>.transx", 0.0f),
		job.get("translate.<xmlattr>.transy", 0.0f));
	unsigned width = job.get("image.<xmlattr>.width", 1920);
	unsigned height = job.get("image.<xmlattr>.height", 1080);
	std::string filename = job.get("image.<xmlattr>.filename", "fractal.png");
	colormap cmap = fromPreset("lightgarden");

	// Generate fractal job
	generate(mbrot, cons, scale, trans, cmap, width, height, filename, mnemonic);
};

// -------------------------------- COMMAND PARSE ---------------------------------

/**
 * The main procedure
 *
 * @param argc the number of command line args
 * @param argv the command line args
 *
 * @return status code
 */
int main(int argc, const char* argv[]) {
	// Soon-to-be user-inputted data
	bool help, cmaps, mbrot;
	float consr, consi, zoom, rotate, transx, transy;
	unsigned width, height;
	std::string xml, cname, filename, mnemonic;

	// Get user input
	po::options_description options("> CUDAFractal [options]");
	options.add_options()
		("help", po::bool_switch(&help), "print help message")
		("cmaps", po::bool_switch(&cmaps), "prints the list of colormap presets")
		("xml", po::value<std::string>(&xml)->default_value(""), "parse xml file")
		("mbrot", po::bool_switch(&mbrot), "compute the mandelbrot fractal algorithm")
		("cr", po::value<float>(&consr)->default_value(-0.4), "real value of c")
		("ci", po::value<float>(&consi)->default_value(0.6), "imaginary value of c")
		("width", po::value<unsigned>(&width)->default_value(1920), "image width")
		("height", po::value<unsigned>(&height)->default_value(1080), "image height")
		("zoom", po::value<float>(&zoom)->default_value(1.0f), "zoom value")
		("rotate", po::value<float>(&rotate)->default_value(0.0f), "rotation value")
		("transx", po::value<float>(&transx)->default_value(0.0f), "x translation")
		("transy", po::value<float>(&transy)->default_value(0.0f), "y translation")
		("cmap", po::value<std::string>(&cname)->default_value("nvidia"), "colormap preset")
		("file", po::value<std::string>(&filename)->default_value("fractal.png"), "output file name")
		("mnemonic", po::value<std::string>(&mnemonic)->default_value("fractal"), "used to identify job");
	po::variables_map vars;
	po::store(po::parse_command_line(argc, argv, options), vars);
	po::notify(vars);

	// Handle different flags
	if (help) {
		std::cout << options << std::endl;
	} else if (!xml.empty()) {

		// Example job
		pt::ptree exampleJob;
		exampleJob.add("<xmlattr>.mnemonic", "exampleFractal");
		exampleJob.add("<xmlattr>.mandelbrot", false);
		exampleJob.add("constant.<xmlattr>.real", -0.8f);
		exampleJob.add("constant.<xmlattr>.imag", 0.9f);
		exampleJob.add("scale.<xmlattr>.rotate", 45.0f);
		exampleJob.add("scale.<xmlattr>.zoom", 0.5f);
		exampleJob.add("translate.<xmlattr>.transx", -0.5);
		exampleJob.add("translate.<xmlattr>.transy", 0.5);
		exampleJob.add("image.<xmlattr>.width", 800);
		exampleJob.add("image.<xmlattr>.height", 800);
		exampleJob.add("image.<xmlattr>.filename", "C:\\Users\\akans\\Desktop\\fractal.png");

		// Do job
		doFractalJob(exampleJob);
	
	} else if (cmaps) {
		listPresets();
	} else {
		// Get colormap and complex values
		colormap cmap = fromPreset(cname);
		hipFloatComplex cons = make_hipFloatComplex(consr, consi);
		hipFloatComplex scale = make_cuScaleComplex(rotate, zoom);
		hipFloatComplex trans = make_hipFloatComplex(transx, transy);

		// Run generator
		generate(mbrot, cons, scale, trans, cmap, width, height, filename, mnemonic);
	}

	return 0;
}

#endif // !__FRACTAL_HOST_CODE__