#include "hip/hip_runtime.h"
#include "lodepng.h"
#include "hip/hip_runtime.h"
#include "hip/hip_complex.h"
#include <iostream>

// PNG Image format
#define IMAGE_NUM_CHANNELS 4
#define IMAGE_RED_CHANNEL 0
#define IMAGE_GREEN_CHANNEL 1
#define IMAGE_BLUE_CHANNEL 2
#define IMAGE_ALPHA_CHANNEL 3

/**
 * Returns complex from the given pixel in the image
 * 
 * @param x the x value of the pixel
 * @param y the y value of the pixel
 * @param w the width of the image
 * @param h the height of the image
 * 
 * @return complex from the given pixel in the image
 */
static __device__ __host__ __inline__
hipFloatComplex fromPixel(unsigned x, unsigned y, unsigned w, unsigned h) {
	return make_hipFloatComplex(
		-2.0 * ((float)w / h) * x / w + ((float)w / h),
		2.0 * y / h - 1.0);
}

/**
 * The iterative process in the julia set. Computes z = z^2 + c 
 * iteratively, with z being initialized to w. Returns the number 
 * of iterations before abs(z) >= 2 (max 255).
 *
 * @param wr real part of w complex
 * @param wi imaginary part of w complex
 * @param cr real part of c complex
 * @param ci imaginary part of c complex
 *
 * @return number of iterations before abs(z) >= 2 (max 255).
 */
static __device__ __host__ __inline__
unsigned char iterations(hipFloatComplex w, hipFloatComplex c) {
	// Set initial z value
	hipFloatComplex z = w;

	// Algorithm
	unsigned char iters;
	for (iters = 0; iters < 255; iters++) {
		// Break if abs(z) >= 2
		if (hipCabsf(z) >= 2) break;

		// Run iteration of z: z = z^2 + c
		z = hipCaddf(hipCmulf(z, z), c);
	}

	// Return iterations
	return iters;
}

/**
 * It assigns the corresponding pixel of the thread to a corresponding starting 
 * complex number z. Then, it runs the juliaset algorithm on z using the given c. 
 * Finally, it computes the color from the resulting iteration number and assigns 
 * that color to the thread's corresponding pixel in the image.
 *
 * @param img_w the width of the image
 * @param img_h the height of the image
 * @param img   the image buffer
 */
__global__
void juliaset(hipFloatComplex c, unsigned w, unsigned h, unsigned char* img) {
	// Get x and y of image (don't run pixels beyond size on img)
	unsigned y = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned x = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= w || y >= h) return;

	// Run iterations algorithm, setting w to the pixel complex
	char iters = iterations(fromPixel(x, y, w, h), c);

	// Append colors to image buffer
	img[(y*w + x)*IMAGE_NUM_CHANNELS + IMAGE_RED_CHANNEL]   = iters; // Red
	img[(y*w + x)*IMAGE_NUM_CHANNELS + IMAGE_GREEN_CHANNEL] = iters; // Green
	img[(y*w + x)*IMAGE_NUM_CHANNELS + IMAGE_BLUE_CHANNEL]  = iters; // Blue
	img[(y*w + x)*IMAGE_NUM_CHANNELS + IMAGE_ALPHA_CHANNEL] = 0xff;  // Alpha
}

/**
 * The main procedure
 *
 * @param argc the number of command line args
 * @param argv the command line args
 *
 * @return status code
 */
int main(int argc, const char* argv[]) {
	// Soon-to-be user-inputted data
	float consr = -0.4;
	float consi = 0.6;
	unsigned width = 1920;
	unsigned height = 1080;
	const char* filename = "C:\\Users\\akans\\Desktop\\fractal.png";

	// Create constant
	hipFloatComplex cons = make_hipFloatComplex(consr, consi);

	// Block space
	// Using 8x8 thread block space because that 
	// divides evenly into most standard resolutions
	int blockSize = 8;
	dim3 blockSpace(blockSize, blockSize);
		
	// Grid space
	// Find the largest side of the image rectangle 
	// and make a square out of that side. Divide 
	// number oftotal "threads" by the block size. 
	// This is the number of the blocks in the grid
	int gridSize = (width >= height ? width : height) / blockSize;
	dim3 gridSpace(gridSize, gridSize);

	// NOTE: 
	//	Investigate why grid spaces or block spaces 
	//	do not work in this case when made rectangular...

	// Create a cuda-managed image buffer and save location at image
	unsigned char* image;
	unsigned length = width*height*IMAGE_NUM_CHANNELS;
	hipMallocManaged(&image, sizeof(unsigned char)*length);

	// Where the magic happens...
	// Call CUDA kernel on the given grid space of blocks
	// Each block being a block space of threads.
	// Each thread computes a separate pixel in the JuliaSet
	std::cout << "Running JuliaSet kernel...";
	juliaset<<<gridSpace, blockSpace>>>(cons, width, height, image);
	hipDeviceSynchronize(); // Wait for kernel to finish
	std::cout << "Done!" << std::endl;

	// Save img buffer to png file
	std::cout << "Saving png...";
	lodepng_encode32_file(filename, image, width, height);
	std::cout << "Done!" << std::endl;
	
	// Free image buffer and exit
	hipFree(image);
	return 0;
}