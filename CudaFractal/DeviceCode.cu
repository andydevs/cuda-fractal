#include "hip/hip_runtime.h"
// Includes
#include "DeviceCode.cuh"

/**
 * Returns complex from the given pixel in the image
 *
 * @param x the x value of the pixel
 * @param y the y value of the pixel
 * @param w the width of the image
 * @param h the height of the image
 * @param s the scale complex
 * @param t the translation complex
 *
 * @return complex from the given pixel in the image
 */
__device__ __host__
hipFloatComplex fromPixel(unsigned x, unsigned y, unsigned w, unsigned h, hipFloatComplex s, hipFloatComplex t) {
	hipFloatComplex z = make_hipFloatComplex(
		((float)2.0) * ((float)w/h) * x/w - ((float)w/h),
		((float)2.0) * y/h - ((float)1.0));

	// Return transform
	return hipCmulf(s, hipCaddf(t, z));
}

/**
 * The iterative process in the julia set. Computes z = z^2 + c
 * iteratively, with z being initialized to w. Returns the number
 * of iterations before abs(z) >= 2 (max 255).
 *
 * @param w complex value w
 * @param c complex value c
 *
 * @return number of iterations before abs(z) >= 2 (max 255).
 */
__device__ __host__ 
unsigned char iterations(hipFloatComplex w, hipFloatComplex c) {
	// Set initial z value
	hipFloatComplex z = w;

	// Algorithm
	unsigned char iters;
	for (iters = 0; iters < 255; iters++) {
		// Break if abs(z) >= 2
		if (hipCabsf(z) >= 2) break;

		// Run iteration of z: z = z^2 + c
		z = hipCaddf(hipCmulf(z, z), c);
	}

	// Return iterations
	return iters;
}