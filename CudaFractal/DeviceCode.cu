#include "hip/hip_runtime.h"
// Includes
#include "Super.h"
#include "DeviceCode.cuh"

// Maximum iterations
#define MAX_ITER 255

/**
 * Returns complex from the given pixel in the image
 *
 * @param x the x value of the pixel
 * @param y the y value of the pixel
 * @param w the width of the image
 * @param h the height of the image
 * @param s the scale complex
 * @param t the translation complex
 *
 * @return complex from the given pixel in the image
 */
__device__
hipFloatComplex fromPixel(unsigned x, unsigned y, unsigned w, unsigned h, hipFloatComplex s, hipFloatComplex t) {
	// z complex value from image pixel
	hipFloatComplex z = make_hipFloatComplex(
		((float)2.0) * ((float)w/h) * x/w - ((float)w/h),
		((float)2.0) * y/h - ((float)1.0));

	// Transform complex value
	return hipCmulf(s, hipCaddf(t, z));
}

/**
 * The iterative process in the julia set. Computes z = z^2 + c
 * iteratively, with z being initialized to w. Returns the number
 * of iterations before abs(z) >= 2 (max 255).
 *
 * @param w complex value w
 * @param c complex value c
 *
 * @return number of iterations before abs(z) >= 2 (max 255).
 */
__device__ 
unsigned char iterations(hipFloatComplex w, hipFloatComplex c) {
	// Set initial z value
	hipFloatComplex z = w;

	// Algorithm
	byte iters;
	for (iters = 0; iters < MAX_ITER; iters++) {
		// Break if abs(z) >= 2
		if (hipCabsf(z) >= 2) break;

		// Run iteration of z: z = z^2 + c
		z = hipCaddf(hipCmulf(z, z), c);
	}

	// Return iterations
	return iters;
}