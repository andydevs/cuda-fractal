#include "hip/hip_runtime.h"
#include "FractalKernels.cuh"
#include "Super.cuh"
#include "DeviceCode.cuh"
#include "Coloring.cuh"
#include <>
#include <hip/hip_runtime.h>

/**
 * It assigns the corresponding pixel of the thread to a corresponding starting
 * complex number z. Then, it runs the juliaset algorithm on z using the given c.
 * Finally, it computes the color from the resulting iteration number and assigns
 * that color to the thread's corresponding pixel in the image.
 *
 * @param c    the complex constant c
 * @param s    the scale complex
 * @param t    the translation complex
 * @param cmap the colormap to use when mapping colors
 * @param w    the width of the image
 * @param h    the height of the image
 * @param img  the image buffer
 */
__global__
void juliaset_kernel(hipFloatComplex c, hipFloatComplex s, hipFloatComplex t, colormap cmap, unsigned w, unsigned h, byte* img) {
	// Get x and y of image (don't run pixels beyond size on img)
	unsigned y = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned x = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= w || y >= h) return;

	// Run iterations algorithm, setting w to the pixel complex, then set pixel in image to mapped color
	setPixel(img, w, h, x, y,
		mapColor(cmap,
			iterations(
				fromPixel(x, y, w, h, s, t), c
			)));
};

/**
 * Launch juliaset kernel
 *
 * It assigns the corresponding pixel of the thread to a corresponding starting
 * complex number z. Then, it runs the juliaset algorithm on z using the given c.
 * Finally, it computes the color from the resulting iteration number and assigns
 * that color to the thread's corresponding pixel in the image.
 *
 * @param c    the complex constant c
 * @param s    the scale complex
 * @param t    the translation complex
 * @param cmap the colormap to use when mapping colors
 * @param w    the width of the image
 * @param h    the height of the image
 * @param img  the image buffer
 */
void juliaset_launcher(hipFloatComplex c, hipFloatComplex s, hipFloatComplex t, colormap cmap, unsigned w, unsigned h, byte* img) {
	// Block space
	// Using 8x8 thread block space because that 
	// divides evenly into most standard resolutions
	int blockSize = 8;
	dim3 blockSpace(blockSize, blockSize);

	// Grid space
	// Find the largest side of the image rectangle
	// and make a square out of that side. Divide 
	// number oftotal "threads" by the block size. 
	// This is the number of the blocks in the grid
	int gridSize = (w >= h ? w : h) / blockSize;
	dim3 gridSpace(gridSize, gridSize);

	// NOTE: 
	//	Investigate why grid spaces or block spaces 
	//	do not work in this case when made rectangular...

	// Launch juliaset kernel
	juliaset_kernel<<<gridSpace, blockSpace>>>(c, s, t, cmap, w, h, img);
};

/**
 * It assigns the corresponding pixel of the thread to a corresponding complex
 * constant number c and sets z to 0. Then, it runs the iteration algorithm on
 * z using the given c.Finally, it computes the color from the resulting iteration
 * number and assigns that color to the thread's corresponding pixel in the image.
 *
 * @param s the scale complex
 * @param t the translation complex
 * @param cmap the colormap to use when mapping colors
 * @param w    the width of the image
 * @param h    the height of the image
 * @param img  the image buffer
 */
__global__
void mandelbrotset_kernel(hipFloatComplex s, hipFloatComplex t, colormap cmap, unsigned w, unsigned h, byte* img) {
	// Get x and y of image (don't run pixels beyond size on img)
	unsigned y = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned x = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= w || y >= h) return;

	// Run iterations algorithm, setting w to 0 and c to the pixel complex, 
	// then set pixel in image to mapped color
	setPixel(img, w, h, x, y,
		mapColor(cmap,
			iterations(
				make_hipFloatComplex(0.0, 0.0),
				fromPixel(x, y, w, h, s, t)
			)));
};

/**
 * Launch mandelbrotset kernel
 *
 * It assigns the corresponding pixel of the thread to a corresponding complex
 * constant number c and sets z to 0. Then, it runs the iteration algorithm on
 * z using the given c.Finally, it computes the color from the resulting iteration
 * number and assigns that color to the thread's corresponding pixel in the image.
 *
 * @param s the scale complex
 * @param t the translation complex
 * @param cmap the colormap to use when mapping colors
 * @param w    the width of the image
 * @param h    the height of the image
 * @param img  the image buffer
 */
void mandelbrotset_launcher(hipFloatComplex s, hipFloatComplex t, colormap cmap, unsigned w, unsigned h, byte* img) {
	// Block space
	// Using 8x8 thread block space because that 
	// divides evenly into most standard resolutions
	int blockSize = 8;
	dim3 blockSpace(blockSize, blockSize);

	// Grid space
	// Find the largest side of the image rectangle
	// and make a square out of that side. Divide 
	// number oftotal "threads" by the block size. 
	// This is the number of the blocks in the grid
	int gridSize = (w >= h ? w : h) / blockSize;
	dim3 gridSpace(gridSize, gridSize);

	// NOTE: 
	//	Investigate why grid spaces or block spaces 
	//	do not work in this case when made rectangular...

	// Run kernel
	mandelbrotset_kernel<<<gridSpace, blockSpace>>>(s, t, cmap, w, h, img);
};